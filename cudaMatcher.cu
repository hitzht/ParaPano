#include "hip/hip_runtime.h"
#include <string>
#include <math.h>
#include <stdio.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "cudaMatcher.h"

#define NUM_THREADS_PER_BLOCK 1024

using namespace std;

CudaMatcher::CudaMatcher() {
    // num_matches = 0;
    cudaDesc1 = NULL;
    cudaDesc2 = NULL;
}

CudaMatcher::~CudaMatcher() {
    if (cudaDesc1) {
        // free image data on CUDA
        hipFree(cudaDesc1);
        hipFree(cudaDesc2);
    }
}

void copyDescriptorToDevice(vector<Descriptor>& desc, Descriptor* cudaDesc) {
    int num_desc = desc.size();
    printf("copyDescriptorToDevice start\n");

    // convert vector to array to be used on Cuda Device
    Descriptor* tempDesc = new Descriptor[num_desc];
    for (int i = 0; i < num_desc; i++) {
        tempDesc[i] = desc[i];
    }

    hipMalloc((void**)&cudaDesc, sizeof(Descriptor) * num_desc);
    hipMemcpy(cudaDesc, tempDesc, sizeof(Descriptor) * num_desc, hipMemcpyHostToDevice);

    delete[] tempDesc;
    printf("copyDescriptorToDevice end\n");
}


void
CudaMatcher::setup(vector<Descriptor>& desc1, vector<Descriptor>& desc2) {
    // printCudaInfo();
    num_desc1 = desc1.size();
    num_desc2 = desc2.size();
    copyDescriptorToDevice(desc1, cudaDesc1);
    copyDescriptorToDevice(desc2, cudaDesc2);
}


__device__ __inline__ int countOneBits(int64_t i) {
    i = i - ((i >> 1) & 0x5555555555555555);
    i = (i & 0x3333333333333333) + ((i >> 2) & 0x3333333333333333);
    return (((i + (i >> 4)) & 0xF0F0F0F0F0F0F0F) * 0x101010101010101) >> 56;
}


__device__ __inline__ int hammingDistance(Descriptor& d1, Descriptor& d2) {
    int dist = 0;
    dist += countOneBits(d1.num0 ^ d2.num0);
    dist += countOneBits(d1.num1 ^ d2.num1);
    dist += countOneBits(d1.num2 ^ d2.num2);
    dist += countOneBits(d1.num3 ^ d2.num3);
    return dist;
}

__device__ void findBestMatch(int idx, Descriptor& d, Descriptor* cudaDesc2, 
    int num_desc2, float* ratios, int* match_indices) {

    int min = INT_MAX;
    int second_min = INT_MAX;
    int min_idx = -1;
    for (int i = 0; i < num_desc2; i++) {
        int dist = hammingDistance(cudaDesc2[i], d);
        if (dist < min) {
            second_min = min;
            min = dist;
            min_idx = i;
        } else if (dist < second_min) {
            second_min = dist;
        }
    }
    float r = second_min == 0 ? 1 : float(min) / second_min;
    match_indices[idx] = min_idx;
    ratios[idx] = r;
}

__global__ void match(Descriptor* cudaDesc1, Descriptor* cudaDesc2, 
    int num_desc1, int num_desc2, float* ratios, int* match_indices) {

    int begin = blockIdx.x * NUM_THREADS_PER_BLOCK + threadIdx.x;
    int end = min(begin + NUM_THREADS_PER_BLOCK, num_desc1);

    for (int i = begin; i < end; i++) {
        findBestMatch(i, cudaDesc1[i], cudaDesc2, num_desc2, 
            ratios, match_indices);
    }
}

void
CudaMatcher::getMatchResult(float* ratios, int* match_indices,
                            float* cuda_ratios, int* cuda_match_indices) {

    hipMemcpy(ratios, cuda_ratios, sizeof(float) * num_desc1,
        hipMemcpyDeviceToHost);

    hipMemcpy(match_indices, cuda_match_indices, sizeof(int) * num_desc1,
        hipMemcpyDeviceToHost);
}

void
CudaMatcher::findMatch() {
    float* ratios;
    float* cuda_ratios;
    int* match_indices;
    int* cuda_match_indices;
    float ratio_threshold = 0.8;

    ratios = new float[num_desc1];
    match_indices = new int[num_desc1];

    hipMalloc(&cuda_ratios, sizeof(float) * num_desc1);
    hipMalloc(&cuda_match_indices, sizeof(int) * num_desc1);
    printf("findMatch: malloc done\n");

    // Spawn CUDA threads
    int num_blocks = (num_desc1 + NUM_THREADS_PER_BLOCK - 1) / NUM_THREADS_PER_BLOCK;

    match<<<num_blocks, NUM_THREADS_PER_BLOCK>>>(cudaDesc1, cudaDesc2, 
        num_desc1, num_desc2, cuda_ratios, cuda_match_indices);
    hipDeviceSynchronize();
    printf("synch done\n");

    getMatchResult(ratios, match_indices, cuda_ratios, cuda_match_indices);

    for (int i = 0; i < num_desc1; i++) {
        if (ratios[i] < ratio_threshold) {
            indices1.push_back(i);
            indices2.push_back(match_indices[i]);
        }
    }
    printf("push back done\n");
    hipFree(cuda_ratios);
    hipFree(cuda_match_indices);
    delete[] ratios;
    delete[] match_indices;
}