#include "hip/hip_runtime.h"
#include <string>
#include <math.h>
#include <stdio.h>
#include <vector>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <driver_functions.h>

#include "cudaMatcher.h"

#define NUM_THREADS_PER_BLOCK 1024

using namespace std;

CudaMatcher::CudaMatcher() {
    // num_matches = 0;
    cudaDesc1 = NULL;
    cudaDesc2 = NULL;
}

CudaMatcher::~CudaMatcher() {
    if (cudaDesc1) {
        // free image data on CUDA
        hipFree(cudaDesc1);
        hipFree(cudaDesc2);
    }
}

void copyDescriptorToDevice(vector<Descriptor>& desc, Descriptor* cudaDesc) {
    int num_desc = desc.size();

    // convert vector to array to be used on Cuda Device
    Descriptor* tempDesc = new Descriptor[num_desc];
    for (int i = 0; i < num_desc; i++) {
        tempDesc[i] = desc[i];
    }

    hipMalloc((void**)&cudaDesc, sizeof(Descriptor) * num_desc);
    hipMemcpy(cudaDesc, tempDesc, sizeof(Descriptor) * num_desc, hipMemcpyHostToDevice);

    delete[] tempDesc;
}


void
CudaMatcher::setup(vector<Descriptor>& desc1, vector<Descriptor>& desc2) {
    // printCudaInfo();
    num_desc1 = desc1.size();
    num_desc2 = desc2.size();
    copyDescriptorToDevice(desc1, cudaDesc1);
    copyDescriptorToDevice(desc2, cudaDesc2);
}


__device__ __inline__ int countOneBits(int64_t x) {
    return 0;
}


__device__ __inline__ int hammingDistance(Descriptor& d1, Descriptor& d2) {
    int dist = 0;
    BitArray& b1 = d1.values;
    BitArray& b2 = d2.values;
    for (int i = 0; i < b1.num_cells; i++) {
        int64_t diff = b1.value[i] ^ b2.value[i];
        dist += countOneBits(diff);
    }
    return dist;
}

__device__ void findBestMatch(int idx, Descriptor& d, Descriptor* cudaDesc2, 
    int num_desc2, float* ratios, int* match_indices) {

    int min = INT_MAX;
    int second_min = INT_MAX;
    int min_idx = -1;
    for (int i = 0; i < num_desc2; i++) {
        int dist = hammingDistance(cudaDesc2[i], d);
        if (dist < min) {
            second_min = min;
            min = dist;
            min_idx = i;
        } else if (dist < second_min) {
            second_min = dist;
        }
    }
    float r = second_min == 0 ? 1 : float(min) / second_min;
    match_indices[idx] = min_idx;
    ratios[idx] = r;
}

__global__ void match(Descriptor* cudaDesc1, Descriptor* cudaDesc2, 
    int num_desc1, int num_desc2, float* ratios, int* match_indices) {

    int begin = blockIdx.x * NUM_THREADS_PER_BLOCK + threadIdx.x;
    int end = min(begin + NUM_THREADS_PER_BLOCK, num_desc1);

    for (int i = begin; i < end; i++) {
        findBestMatch(i, cudaDesc1[i], cudaDesc2, num_desc2, 
            ratios, match_indices);
    }
}

void
CudaMatcher::findMatch() {
    float* ratios;
    int* match_indices;
    float ratio_threshold = 0.8;

    hipMalloc(&ratios, sizeof(float) * num_desc1);
    hipMalloc(&match_indices, sizeof(int) * num_desc1);

    // Spawn CUDA threads
    int num_blocks = (num_desc1 + NUM_THREADS_PER_BLOCK - 1) / NUM_THREADS_PER_BLOCK;

    match<<<num_blocks, NUM_THREADS_PER_BLOCK>>>(cudaDesc1, cudaDesc2, 
        num_desc1, num_desc2, ratios, match_indices);
    hipDeviceSynchronize();

    for (int i = 0; i < num_desc1; i++) {
        if (ratios[i] < ratio_threshold) {
            indices1.push_back(i);
            indices2.push_back(match_indices[i]);
        }
    }
    hipFree(ratios);
    hipFree(match_indices);
}